
#include<stdio.h>
#include<hip/hip_runtime.h>

__global__
void deviceKernel(int *a, int N)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = idx; i < N; i += stride)
  {
    a[i] = 1;
  }
}

void hostFunction(int *a, int N)
{
  for (int i = 0; i < N; ++i)
  {
    a[i] = 1;
  }
}

int main()
{

  int N = 2<<24;
  size_t size = N * sizeof(int);
  int *a;
  hipMallocManaged(&a, size);

int threadsPerBlock = 32;
int blocksPerGrid =((N + threadsPerBlock - 1)/ threadsPerBlock);


deviceKernel<<<blocksPerGrid,threadsPerBlock>>>(a,N);
hipDeviceSynchronize();

hostFunction(a,N);

  /*
   * Conduct experiments to learn more about the behavior of
   * `cudaMallocManaged`.
   *
   * What happens when unified memory is accessed only by the GPU?
   * What happens when unified memory is accessed only by the CPU?
   * What happens when unified memory is accessed first by the GPU then the CPU?
   * What happens when unified memory is accessed first by the CPU then the GPU?
   *
   * Hypothesize about UM behavior, page faulting specificially, before each
   * experiement, and then verify by running `nvprof`.
   
    for (int i = 0; i < N; ++i)
    {
        if (a[i]!=1)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED\n");
 */
  hipFree(a);
}

